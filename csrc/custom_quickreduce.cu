#include "hip/hip_runtime.h"
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/all.h>

#ifdef USE_ROCM

  #include "quickreduce/quick_reduce.h"

quickreduce::fptr_t init_custom_qr(int64_t rank, int64_t world_size) {
  if (world_size > 8)
    throw std::invalid_argument("world size > 8 is not supported");
  if (world_size % 2 != 0)
    throw std::invalid_argument("Odd num gpus is not supported for now");
  if (rank < 0 || rank >= world_size)
    throw std::invalid_argument("invalid rank passed in");
  quickreduce::DeviceComms* fptr = new quickreduce::DeviceComms();
  fptr->init(world_size, rank);
  return (quickreduce::fptr_t)fptr;
}

void qr_destroy(quickreduce::fptr_t _fa) {
  auto fa = reinterpret_cast<quickreduce::DeviceComms*>(_fa);
  fa->destroy();
  delete fa;
}

torch::Tensor qr_get_handle(quickreduce::fptr_t _fa) {
  auto fa = reinterpret_cast<quickreduce::DeviceComms*>(_fa);
  hipIpcMemHandle_t handle = fa->get_handle();
  auto device_index = c10::cuda::current_device();
  auto options =
      torch::TensorOptions().dtype(torch::kUInt8).device(torch::kCPU);
  auto data_handle =
      torch::empty({static_cast<int64_t>(sizeof(hipIpcMemHandle_t))}, options);
  std::memcpy(data_handle.data_ptr(), &handle, sizeof(hipIpcMemHandle_t));
  return data_handle;
}

void qr_open_handles(quickreduce::fptr_t _fa,
                     const std::vector<torch::Tensor>& handles) {
  auto fa = reinterpret_cast<quickreduce::DeviceComms*>(_fa);
  std::vector<hipIpcMemHandle_t> ipc_handles;
  ipc_handles.reserve(handles.size());
  for (auto& handle : handles) {
    // Ensure the tensor is on the same device as the current device.
    hipIpcMemHandle_t ipc_handle;
    std::memcpy(&ipc_handle, handle.data_ptr(), sizeof(hipIpcMemHandle_t));
    ipc_handles.push_back(ipc_handle);
  }
  fa->open_ipc_handles(ipc_handles);
}

void qr_all_reduce(quickreduce::fptr_t _fa, torch::Tensor& inp,
                   torch::Tensor& out, int64_t algo_int) {
  auto fa = reinterpret_cast<quickreduce::DeviceComms*>(_fa);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(inp));
  auto stream = at::cuda::getCurrentHIPStreamMasqueradingAsCUDA();

  TORCH_CHECK_EQ(inp.scalar_type(), out.scalar_type());
  TORCH_CHECK_EQ(inp.numel(), out.numel());

  auto algo = static_cast<quickreduce::QuickReduceAlgo>(algo_int);
  if (out.scalar_type() == at::ScalarType::Half) {
    fa->allreduce<half>(algo_int, stream,
                        reinterpret_cast<half*>(inp.data_ptr()),
                        reinterpret_cast<half*>(out.data_ptr()), out.numel());
  } else if (out.scalar_type() == at::ScalarType::BFloat16) {
    fa->allreduce<quickreduce::hip_bfloat16>(
        algo_int, stream,
        reinterpret_cast<quickreduce::hip_bfloat16*>(inp.data_ptr()),
        reinterpret_cast<quickreduce::hip_bfloat16*>(out.data_ptr()),
        out.numel());
  } else {
    throw std::runtime_error(
        "quick allreduce only supports float16 and bfloat16");
  }
}

int64_t qr_max_size() {
  return static_cast<int64_t>(quickreduce::DeviceComms::kMaxProblemSize);
}

int64_t qr_min_size() {
  return static_cast<int64_t>(quickreduce::kBlockSize * quickreduce::kAtoms *
                              sizeof(quickreduce::int32x4_t));
}

#endif  // USE_ROCM