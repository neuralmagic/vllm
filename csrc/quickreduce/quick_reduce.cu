
#include <hip/hip_runtime.h>
#ifdef USE_ROCM

  #include <hip/hip_runtime.h>

  #include "quick_reduce_impl.cuh"
  #include "quick_reduce.h"

void DeviceComms::init(int world_size, int rank) {
  destroy();
  this->world_size = world_size;
  this->rank = rank;

  // Allocate buffer size for worst case: Twoshot FP16 2-stage buffer.
  long flags_buffer_size = 2 * world_size * kMaxTiles * sizeof(int);
  long data_buffer_size = 2 * kMaxProblemSize;
  long total_buffer_size = flags_buffer_size + data_buffer_size;
  data_offset = flags_buffer_size;
  HIP_CHECK(hipExtMallocWithFlags((void**)&dbuffer, total_buffer_size,
                                  hipDeviceMallocUncached));

  // Clear the flags buffer.
  HIP_CHECK(hipMemset(dbuffer, 0, flags_buffer_size));

  // Device-side list of IPC buffers.
  buffer_list.resize(world_size);
  HIP_CHECK(hipMalloc(&dbuffer_list, world_size * sizeof(uint8_t*)));

  // Create IPC handles for rank's communication buffer.
  all_buffer_ipc_handles.resize(world_size);
  HIP_CHECK(hipIpcGetMemHandle(&buffer_ipc_handle, dbuffer));

  initialized = true;
}

void DeviceComms::destroy() {
  if (initialized) {
    for (int i = 0; i < world_size; i++) {
      if (i != rank) {
        HIP_CHECK(hipIpcCloseMemHandle(dbuffer_list[i]));
      }
    }

    HIP_CHECK(hipFree(dbuffer));
    HIP_CHECK(hipFree(dbuffer_list));

    initialized = false;
  }
}

void DeviceComms::open_ipc_handles(
    std::vector<hipIpcMemHandle_t> const& ipc_handles) {
  assert(ipc_handles.size() == all_buffer_ipc_handles.size());
  for (int i = 0; i < world_size; i++) {
    all_buffer_ipc_handles[i] = ipc_handles[i];
  }

  // Open device memory access to the IPC communication buffers.
  // Note: For our own rank, we do not need to open a handle.
  for (int i = 0; i < world_size; i++) {
    if (i != rank) {
      HIP_CHECK(hipIpcOpenMemHandle((void**)&buffer_list[i],
                                    all_buffer_ipc_handles[i],
                                    hipIpcMemLazyEnablePeerAccess));
    } else {
      buffer_list[i] = dbuffer;
    }
  }

  HIP_CHECK(hipMemcpy(dbuffer_list, buffer_list.data(),
                      world_size * sizeof(uint8_t*), hipMemcpyHostToDevice));
}

// ============================================================
// KERNEL
// ============================================================
template <typename AllReduceKenel>
__global__ __quickreduce_launch_bounds__ static void allreduce_prototype(
    half const* A, half* B, int N, int num_blocks, int world_size, int rank,
    uint8_t** dbuffer_list, long data_offset, int flag_color) {
  int block = blockIdx.x;
  int grid = gridDim.x;

  while (block < num_blocks) {
    AllReduceKenel::run(A, B, N, block, num_blocks, world_size, rank,
                        dbuffer_list, data_offset, flag_color);
    block += grid;
  }
}

  // ============================================================
  // DISPATCH
  // ============================================================
  #define TWOSHOT_DISPATCH(__codec)                                          \
    if (world_size == 2) {                                                   \
      using LineCodec = __codec<2>;                                          \
      using AllReduceKernel = AllReduceTwoshot<LineCodec>;                   \
      hipLaunchKernelGGL((allreduce_prototype<AllReduceKernel>), dim3(grid), \
                         dim3(kBlock), 0, stream, A, B, N, num_blocks,       \
                         world_size, rank, dbuffer_list, data_offset,        \
                         flag_color);                                        \
    } else if (world_size == 4) {                                            \
      using LineCodec = __codec<4>;                                          \
      using AllReduceKernel = AllReduceTwoshot<LineCodec>;                   \
      hipLaunchKernelGGL((allreduce_prototype<AllReduceKernel>), dim3(grid), \
                         dim3(kBlock), 0, stream, A, B, N, num_blocks,       \
                         world_size, rank, dbuffer_list, data_offset,        \
                         flag_color);                                        \
    } else if (world_size == 8) {                                            \
      using LineCodec = __codec<8>;                                          \
      using AllReduceKernel = AllReduceTwoshot<LineCodec>;                   \
      hipLaunchKernelGGL((allreduce_prototype<AllReduceKernel>), dim3(grid), \
                         dim3(kBlock), 0, stream, A, B, N, num_blocks,       \
                         world_size, rank, dbuffer_list, data_offset,        \
                         flag_color);                                        \
    }

void DeviceComms::allreduce(int profile, hipStream_t stream, half const* A,
                            half* B, int N) {
  if (world_size != 2 && world_size != 4 && world_size != 8) {
    throw std::runtime_error("All Reduce not supported for world_size = " +
                             std::to_string(world_size));
  }

  // Configuration.
  long msg_size = N * sizeof(half);
  unsigned long num_blocks = divceil(msg_size, kTileSize);
  unsigned long grid = min(304 * 4, num_blocks);
  // -------------------------------------------------
  // All reduce dispatch.
  QuickReduceProfile dprofile = static_cast<QuickReduceProfile>(profile);

  switch (dprofile) {
    case QuickReduceProfile::ONESHOT_FP16:
      using AllReduceKernel = AllReduceOneshot;
      hipLaunchKernelGGL((allreduce_prototype<AllReduceKernel>), dim3(grid),
                         dim3(kBlock), 0, stream, A, B, N, num_blocks,
                         world_size, rank, dbuffer_list, data_offset,
                         flag_color);
      break;
    case QuickReduceProfile::TWOSHOT_FP8:
      throw std::runtime_error("FP8 is not supported");
      // TWOSHOT_DISPATCH(TwoshotFP8LineCodec)
      break;
    case QuickReduceProfile::TWOSHOT_Q8:
      TWOSHOT_DISPATCH(TwoshotQ8LineCodec)
      break;
    case QuickReduceProfile::TWOSHOT_MAX_MIN_Q8:
      TWOSHOT_DISPATCH(TwoshotMaxMinQ8LineCodec)
      break;
    case QuickReduceProfile::TWOSHOT_Q6:
      TWOSHOT_DISPATCH(TwoshotQ6LineCodec)
      break;
    case QuickReduceProfile::TWOSHOT_Q4:
      TWOSHOT_DISPATCH(TwoshotQ4LineCodec)
      break;
    default:
      TWOSHOT_DISPATCH(TwoshotFP16LineCodec)
      break;
  }
  HIP_CHECK(cudaGetLastError());

  // -------------------------------------------------
  // Rotate the flag color.
  flag_color++;
}

#endif  // USE_ROCM